#include "hip/hip_runtime.h"
#include "api.hpp"
#include <iostream>
#include <fstream>
#include <time.h>

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__)


void check_cuda(hipError_t result, char const *const func, const char *const file, int const line)
{
    if(result)
    {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << "at" << 
            file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}



void render(nanovdb::GridHandle<nanovdb::CudaDeviceBuffer>& handle,
            const RenderSetting& set_info)
{
    auto* d_grid = handle.deviceGrid();
    auto* h_grid = handle.grid<float>();
    auto acc = h_grid->getAccessor();
    auto bbox = h_grid->indexBBox();
    
    // check max density
    float max_density = searchMaxDensity(bbox, acc);

    //get data from render setting
    int nx = set_info.width;
    int ny = set_info.height;
    int tx = 8;
    int ty = 8;
    dim3 blocks(nx/tx + 1, ny/ty + 1);
    dim3 threads(tx, ty);

    //init random state per pixel
    hiprandState *d_rand_state;
    checkCudaErrors(hipMalloc((void **)&d_rand_state, nx * ny * sizeof(hiprandState)));
    random_init<<<blocks, threads>>>(nx, ny, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    std::cerr << "Rendering a " << nx << "x" << ny << " image ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    int num_pixels = nx*ny;
    size_t fb_size = 3*num_pixels*sizeof(float);

    //malloc pixel buffer(for unified memory)
    float* fb;
    checkCudaErrors(hipMallocManaged((void**)&fb, fb_size));

    //start time
    clock_t start, stop;
    start = clock();

    //call kernel function
    renderKernel<<<blocks, threads>>>(fb, nx, ny);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    //finish time
    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "took " << timer_seconds << " seconds.\n";

    //save ppm
    SavePPM(fb, nx, ny);

    //free several data
    checkCudaErrors(hipFree(fb));
}